#include "hip/hip_runtime.h"
#include "cuda_kernels.h"

inline __device__ __host__ size_t div_ceil(size_t a, size_t b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


extern "C" __global__ void convert_dev_floats(int N, float * float_ptr, __half * half_ptr){

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N){
		half_ptr[i] = __float2half(float_ptr[i]);
	}
}


extern "C" __global__ void add_fp16_kernel(size_t N, __half * d_A, __half * d_B, __half * d_out) {

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N){
		d_out[i] = d_A[i] + d_B[i];
	}
}

// should launch with N / 2
extern "C" __global__ void add_fp16_vec_kernel(size_t N, __half2 * d_A, __half2 * d_B, __half2 * d_out) {

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N / 2){
		d_out[i] = d_A[i] + d_B[i];
	}
}

extern "C" __global__ void add_fp32_kernel(size_t N, float * d_A, float * d_B, float * d_out) {

	size_t i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N){
		d_out[i] = d_A[i] + d_B[i];
	}
}

extern "C" __global__ void naive_matmul_fp16_kernel(int M, int K, int N, float alpha, __half *A, __half *B, float beta, __half *C) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < M && y < N){
		float temp = 0;
		for (int i = 0; i < K; i++){
			temp += __half2float(A[x * K + i]) * __half2float(B[i * N + y]);
		}
		C[x * N + y] = __float2half(alpha * temp + beta * __half2float(C[x * N + y]));
	}
}

// REFERENCE / CREDIT...
// FROM: https://github.com/Bruce-Lee-LY/cuda_hgemm/blob/master/src/mma/mma_async_stage4.cu

#define MMA_M 16
#define MMA_N 8
#define MMA_K 16

#define BLOCK_ROWS 256
#define BLOCK_COLS 128

#define WARP_ROWS 64
#define WARP_COLS 64

#define BLOCK_ROW_WARPS 2  // BLOCK_COLS / WARP_COLS
#define BLOCK_COL_WARPS 4  // BLOCK_ROWS / WARP_ROWS

#define BLOCK_ROW_TILES 16  // BLOCK_COLS / MMA_N
#define BLOCK_COL_TILES 16  // BLOCK_ROWS / MMA_M

#define WARP_ROW_TILES 8  // WARP_COLS / MMA_N
#define WARP_COL_TILES 4  // WARP_ROWS / MMA_M

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8      // BLOCK_ROW_WARPS * BLOCK_COL_WARPS
#define THREADS_PER_BLOCK 256  // WARP_SIZE * WARPS_PER_BLOCK

#define CHUNK_K 2  // 32 / MMA_K

#define THREAD_COPY_BYTES 16

#define CHUNK_LINE_BYTES 64          // CHUNK_K * MMA_K * sizeof(half)
#define CHUNK_COPY_LINES_PER_WARP 8  // WARP_SIZE * THREAD_COPY_BYTES / CHUNK_LINE_BYTES
#define CHUNK_COPY_LINE_LANES 4      // WARP_SIZE / CHUNK_COPY_LINES_PER_WARP

#define AB_SMEM_STRIDE 32  // CHUNK_K * MMA_K

#define C_SMEM_STRIDE 128  // BLOCK_COLS
#define C_SMEM_OFFSET 64   // WARP_COLS

#define BLOCK_STRIDE 16

#define SMEM_BANK_ROWS 2  // 32 * 4 / (AB_SMEM_STRIDE * sizeof(half))

#define PERMUTED_OFFSET 8
#define PERMUTED_COLS 4

#define K_STAGE 4

extern "C" __global__ void matmul_fp16_kernel(const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C,
                                     size_t M, size_t N, size_t K, float alpha, float beta) {
    const size_t M_tiles = div_ceil(M, MMA_M);
    const size_t N_tiles = div_ceil(N, MMA_N);
    const size_t K_tiles = div_ceil(K, MMA_K);

    const size_t block_tile_i =
        (blockIdx.z % 2) ? ((gridDim.y - blockIdx.y - 1) * BLOCK_COL_TILES) : (blockIdx.y * BLOCK_COL_TILES);
    const size_t block_tile_j = (blockIdx.z * gridDim.x + blockIdx.x) * BLOCK_ROW_TILES;

    if (block_tile_i >= M_tiles || block_tile_j >= N_tiles) {
        return;
    }

    extern __shared__ half smem[][AB_SMEM_STRIDE];

    const size_t warp_id = threadIdx.x / WARP_SIZE;
    const size_t lane_id = threadIdx.x % WARP_SIZE;

    constexpr size_t B_smem_idx_off = BLOCK_ROWS;
    constexpr size_t smem_stage_off = BLOCK_ROWS + BLOCK_COLS;

    half *smem_warp_tile_row_ptr = &smem[0][0] + (warp_id / BLOCK_ROW_WARPS) * C_SMEM_STRIDE * WARP_ROWS;
    const half *smem_warp_stream_ptr = &smem[0][0] + warp_id * MMA_M * 2 * C_SMEM_STRIDE;

    const size_t gmem_idx = (block_tile_i + warp_id * 2) * MMA_M * N + block_tile_j * MMA_N;
    const half *src_gmem_warp_stream_ptr = &C[gmem_idx];

    uint32_t RC[WARP_COL_TILES][WARP_ROW_TILES][2];

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            RC[i][j][0] = 0;
            RC[i][j][1] = 0;
        }
    }

    const half *A_warp_ptr = &A[block_tile_i * MMA_M * K] + BLOCK_ROWS / WARPS_PER_BLOCK * K * warp_id;
    const half *B_warp_ptr = &B[block_tile_j * MMA_N * K] + BLOCK_COLS / WARPS_PER_BLOCK * K * warp_id;

    constexpr size_t A_smem_iters = BLOCK_ROWS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);
    constexpr size_t B_smem_iters = BLOCK_COLS / (CHUNK_COPY_LINES_PER_WARP * WARPS_PER_BLOCK);

    size_t smem_store_idx = 0;
    size_t smem_load_idx = 0;

    size_t smem_store_off = 0;
    size_t smem_load_off = 0;

    size_t A_smem_idx = 0;
    int4 *A_lane_ptr = nullptr;

    size_t B_smem_idx = 0;
    int4 *B_lane_ptr = nullptr;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4 *)(A_warp_ptr + (lane_id / CHUNK_COPY_LINE_LANES) * K) + (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off + BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4 *)(B_warp_ptr + (lane_id / CHUNK_COPY_LINE_LANES) * K) + (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();

    smem_store_idx = (smem_store_idx + 1) % K_STAGE;
    smem_store_off = smem_store_idx * smem_stage_off;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4 *)(A_warp_ptr + CHUNK_K * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off + BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4 *)(B_warp_ptr + CHUNK_K * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();

    smem_store_idx = (smem_store_idx + 1) % K_STAGE;
    smem_store_off = smem_store_idx * smem_stage_off;

    A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
    A_lane_ptr = (int4 *)(A_warp_ptr + 2 * CHUNK_K * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < A_smem_iters; ++i) {
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

        A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    B_smem_idx = smem_store_off + B_smem_idx_off + BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
    B_lane_ptr = (int4 *)(B_warp_ptr + 2 * CHUNK_K * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                 (lane_id % CHUNK_COPY_LINE_LANES);
    B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
    for (size_t i = 0; i < B_smem_iters; ++i) {
        uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                                    ((lane_id % CHUNK_COPY_LINE_LANES +
                                      (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                     CHUNK_COPY_LINE_LANES) *
                                        THREAD_COPY_BYTES;

        CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

        B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
        B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
    }

    CP_ASYNC_COMMIT_GROUP();
    CP_ASYNC_WAIT_GROUP(2);

    __syncthreads();

    uint32_t RA[2][WARP_COL_TILES][4];
    uint32_t RB[2][WARP_ROW_TILES][2];

    size_t reg_store_idx = 0;
    size_t reg_load_idx = 1;

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
        size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
        uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
            &smem[A_smem_idx + lane_id % 16][((lane_id / 16) * 8 + (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                                                                       SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                                             AB_SMEM_STRIDE]);

        LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2], RA[reg_store_idx][i][3],
                    A_smem_lane_addr);
    }

#pragma unroll
    for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
        size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
        uint32_t B_smem_lane_addr =
            __cvta_generic_to_shared(&smem[B_smem_idx + lane_id % 8]
                                          [(((lane_id / 8) % 2) * 8 + (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                                                                          SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                                           AB_SMEM_STRIDE]);

        LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
    }

#pragma unroll
    for (size_t tile_k = CHUNK_K * (K_STAGE - 1); tile_k < K_tiles; tile_k += CHUNK_K) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                        RA[reg_store_idx][i][3], A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0], RA[reg_load_idx][i][1],
                          RA[reg_load_idx][i][2], RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
            }
        }

        smem_store_idx = (smem_store_idx + 1) % K_STAGE;
        smem_store_off = smem_store_idx * smem_stage_off;

        A_smem_idx = smem_store_off + BLOCK_ROWS / WARPS_PER_BLOCK * warp_id;
        A_lane_ptr = (int4 *)(A_warp_ptr + tile_k * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                     (lane_id % CHUNK_COPY_LINE_LANES);
        A_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < A_smem_iters / CHUNK_K; ++i) {
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                                        ((lane_id % CHUNK_COPY_LINE_LANES +
                                          (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                         CHUNK_COPY_LINE_LANES) *
                                            THREAD_COPY_BYTES;

            CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

            A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        B_smem_idx = smem_store_off + B_smem_idx_off + BLOCK_COLS / WARPS_PER_BLOCK * warp_id;
        B_lane_ptr = (int4 *)(B_warp_ptr + tile_k * MMA_K + (lane_id / CHUNK_COPY_LINE_LANES) * K) +
                     (lane_id % CHUNK_COPY_LINE_LANES);
        B_smem_idx += lane_id / CHUNK_COPY_LINE_LANES;

#pragma unroll
        for (size_t i = 0; i < B_smem_iters / CHUNK_K; ++i) {
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                                        ((lane_id % CHUNK_COPY_LINE_LANES +
                                          (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                         CHUNK_COPY_LINE_LANES) *
                                            THREAD_COPY_BYTES;

            CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

            B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        smem_load_idx = (smem_load_idx + 1) % K_STAGE;
        smem_load_off = smem_load_idx * smem_stage_off;

#pragma unroll
        for (size_t i = (CHUNK_K - 1) * A_smem_iters / CHUNK_K; i < A_smem_iters; ++i) {
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(&smem[A_smem_idx][0]) +
                                        ((lane_id % CHUNK_COPY_LINE_LANES +
                                          (A_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                         CHUNK_COPY_LINE_LANES) *
                                            THREAD_COPY_BYTES;

            CP_ASYNC_CG(A_smem_lane_addr, A_lane_ptr, THREAD_COPY_BYTES);

            A_lane_ptr = (int4 *)((half *)A_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            A_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

#pragma unroll
        for (size_t i = (CHUNK_K - 1) * B_smem_iters / CHUNK_K; i < B_smem_iters; ++i) {
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(&smem[B_smem_idx][0]) +
                                        ((lane_id % CHUNK_COPY_LINE_LANES +
                                          (B_smem_idx % (CHUNK_COPY_LINE_LANES * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS) %
                                         CHUNK_COPY_LINE_LANES) *
                                            THREAD_COPY_BYTES;

            CP_ASYNC_CG(B_smem_lane_addr, B_lane_ptr, THREAD_COPY_BYTES);

            B_lane_ptr = (int4 *)((half *)B_lane_ptr + CHUNK_COPY_LINES_PER_WARP * K);
            B_smem_idx += CHUNK_COPY_LINES_PER_WARP;
        }

        CP_ASYNC_COMMIT_GROUP();
        CP_ASYNC_WAIT_GROUP(2);

        __syncthreads();

        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
            uint32_t A_smem_lane_addr =
                __cvta_generic_to_shared(&smem[A_smem_idx + lane_id % 16]
                                              [((lane_id / 16) * 8 + (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) /
                                                                         SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                                               AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                        RA[reg_store_idx][i][3], A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0], RA[reg_load_idx][i][1],
                          RA[reg_load_idx][i][2], RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
            }
        }
    }

#pragma unroll
    for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                        RA[reg_store_idx][i][3], A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0], RA[reg_load_idx][i][1],
                          RA[reg_load_idx][i][2], RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
            }
        }

        if (k_step + 2 == CHUNK_K) {
            smem_load_idx = (smem_load_idx + 1) % K_STAGE;
            smem_load_off = smem_load_idx * smem_stage_off;

            CP_ASYNC_WAIT_GROUP(1);

            __syncthreads();
        }
    }

#pragma unroll
    for (size_t k_step = 0; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                        RA[reg_store_idx][i][3], A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(((k_step + 1) % CHUNK_K) * MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0], RA[reg_load_idx][i][1],
                          RA[reg_load_idx][i][2], RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
            }
        }

        if (k_step + 2 == CHUNK_K) {
            smem_load_idx = (smem_load_idx + 1) % K_STAGE;
            smem_load_off = smem_load_idx * smem_stage_off;

            CP_ASYNC_WAIT_GROUP(0);

            __syncthreads();
        }
    }

#pragma unroll
    for (size_t k_step = 1; k_step < CHUNK_K; ++k_step) {
        reg_store_idx ^= 1;
        reg_load_idx ^= 1;

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
            size_t A_smem_idx = smem_load_off + (warp_id / BLOCK_ROW_WARPS) * WARP_ROWS + i * MMA_M;
            uint32_t A_smem_lane_addr = __cvta_generic_to_shared(
                &smem[A_smem_idx + lane_id % 16]
                     [(k_step * MMA_K + (lane_id / 16) * 8 +
                       (lane_id % 16 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X4(RA[reg_store_idx][i][0], RA[reg_store_idx][i][1], RA[reg_store_idx][i][2],
                        RA[reg_store_idx][i][3], A_smem_lane_addr);
        }

#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t B_smem_idx = smem_load_off + B_smem_idx_off + (warp_id % BLOCK_ROW_WARPS) * WARP_COLS + j * MMA_N;
            uint32_t B_smem_lane_addr = __cvta_generic_to_shared(
                &smem[B_smem_idx + lane_id % 8]
                     [(k_step * MMA_K + ((lane_id / 8) % 2) * 8 +
                       (lane_id % 8 % (PERMUTED_COLS * SMEM_BANK_ROWS)) / SMEM_BANK_ROWS * PERMUTED_OFFSET) %
                      AB_SMEM_STRIDE]);

            LDMATRIX_X2(RB[reg_store_idx][j][0], RB[reg_store_idx][j][1], B_smem_lane_addr);
        }

#pragma unroll
        for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
            for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
                size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

                HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_load_idx][i][0], RA[reg_load_idx][i][1],
                          RA[reg_load_idx][i][2], RA[reg_load_idx][i][3], RB[reg_load_idx][j_s][0],
                          RB[reg_load_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
            }
        }
    }

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            size_t j_s = (i % 2) ? (WARP_ROW_TILES - j - 1) : j;

            HMMA16816(RC[i][j_s][0], RC[i][j_s][1], RA[reg_store_idx][i][0], RA[reg_store_idx][i][1],
                      RA[reg_store_idx][i][2], RA[reg_store_idx][i][3], RB[reg_store_idx][j_s][0],
                      RB[reg_store_idx][j_s][1], RC[i][j_s][0], RC[i][j_s][1]);
        }
    }

    __syncthreads();

#pragma unroll
    for (size_t i = 0; i < WARP_COL_TILES; ++i) {
#pragma unroll
        for (size_t j = 0; j < WARP_ROW_TILES; ++j) {
            half *lane_ptr0 =
                smem_warp_tile_row_ptr + (i * MMA_M + lane_id / 4) * C_SMEM_STRIDE +
                ((warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                 (lane_id % 4) * sizeof(uint32_t) / sizeof(half) + ((lane_id / 4) % 8) * PERMUTED_OFFSET) %
                    C_SMEM_STRIDE;
            half *lane_ptr1 =
                smem_warp_tile_row_ptr + (i * MMA_M + lane_id / 4 + 8) * C_SMEM_STRIDE +
                ((warp_id % BLOCK_ROW_WARPS) * C_SMEM_OFFSET + j * MMA_N +
                 (lane_id % 4) * sizeof(uint32_t) / sizeof(half) + ((lane_id / 4 + 8) % 8) * PERMUTED_OFFSET) %
                    C_SMEM_STRIDE;

            *((__half2 *)(lane_ptr0)) = __hmul2(__float2half2_rn(alpha), *((__half2 *) &RC[i][j][0]));
            *((__half2 *)(lane_ptr1)) = __hmul2(__float2half2_rn(alpha), *((__half2 *) &RC[i][j][1]));
            //*((uint32_t *)(lane_ptr0)) = RC[i][j][0];
            //*((uint32_t *)(lane_ptr1)) = RC[i][j][1];
        }
    }

    __syncthreads();

#pragma unroll
    for (size_t i = 0; i < MMA_M; ++i) {
        /*
        *((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) =
            *((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) +
              (lane_id % 16 + (i * 2 + lane_id / 16) % 8) % (C_SMEM_STRIDE * sizeof(half) / THREAD_COPY_BYTES));
    	*/
        
        if ((i * 2 + lane_id / 16) < M){
        	*((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16)) = __hmul2(__float2half2_rn(beta), *((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16))) + *((__half2 *)((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) + (lane_id % 16 + (i * 2 + lane_id / 16) % 8) % (C_SMEM_STRIDE * sizeof(half) / THREAD_COPY_BYTES)));
        	*((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 1) = __hmul2(__float2half2_rn(beta), *((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 1)) + *((__half2 *)((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) + (lane_id % 16 + (i * 2 + lane_id / 16) % 8) % (C_SMEM_STRIDE * sizeof(half) / THREAD_COPY_BYTES)) + 1);
        	*((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 2) = __hmul2(__float2half2_rn(beta), *((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 2)) + *((__half2 *)((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) + (lane_id % 16 + (i * 2 + lane_id / 16) % 8) % (C_SMEM_STRIDE * sizeof(half) / THREAD_COPY_BYTES)) + 2);
        	*((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 3) = __hmul2(__float2half2_rn(beta), *((__half2 *)((int4 *)(src_gmem_warp_stream_ptr + (i * 2 + lane_id / 16) * N) + lane_id % 16) + 3)) + *((__half2 *)((int4 *)(smem_warp_stream_ptr + (i * 2 + lane_id / 16) * C_SMEM_STRIDE) + (lane_id % 16 + (i * 2 + lane_id / 16) % 8) % (C_SMEM_STRIDE * sizeof(half) / THREAD_COPY_BYTES)) + 3);
    	}
    }
}



// Out-of-place Transpose!

// Tile dim = 32
// Block rows = 8

// Launch config:

// gridDim.x = ceil(n_orig_rows / 32)
// gridDim.y = ceil(n_orig_cols / 32)

// blockDim.x = 8 * 32
extern "C" __global__ void transpose_fp16_kernel(int n_orig_rows, int n_orig_cols, const __half * __restrict__ in, __half * __restrict__ out) {
	
	// +1 to avoid bank conflict
	__shared__ __half tile[32][32 + 1];
	
	// every thread block will do 32x32 square
	// but within thread block each thread will do 
	// 4 items

	// block_size = 32
	int block_row_start = blockIdx.x * 32;
	int block_col_start = blockIdx.y * 32;

	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	uint64_t total_size = n_orig_rows * n_orig_cols;

	// if this thread's colu

	// Read from original

	// each thread writes to elements from 4 rows of original
	// into tile and stores them

	// contiguous threads (thread_id's in block) will read contiguous data in this way
	// for good coalescing

	

	// each iteration warp loads a row

	uint64_t cur_ind;

	#pragma unroll
	for (int j = 0; j < 32; j += 8){
		cur_ind = (block_row_start + (warp_id+j))*n_orig_cols + (block_col_start + lane_id);
		if (cur_ind < total_size){
			tile[warp_id+j][lane_id] = in[cur_ind];
		}
	}

	__syncthreads();
	
	// now load the item in tile into transposed memory location
	#pragma unroll

	// each iteration warp stores a column of original, now a row in transposed
	for (int j = 0; j < 32; j += 8){
		cur_ind = (block_col_start +(warp_id+j))*n_orig_rows + (block_row_start + lane_id);
		if (cur_ind < total_size){
			out[cur_ind] = tile[lane_id][warp_id + j];
		}
	}
}

// num_stages is defined by amount of smem avail, so needs to be passed in as arg
extern "C" __global__ void rms_norm_fp16_kernel(float eps, int n_rows, int n_cols, __half * rms_weight, __half * X, __half * out, float * sq_sums) {

	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];

	__half * row = (__half *) sdata;
	__half * weights = row + n_cols;

	// every warp will have a reduced value
	__shared__ float reduction_data[32];

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	// Load weights which are shared between all rows (when doing output in item 3...)
	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weights[i] = rms_weight[i];
	}
	__syncthreads();

	__half cur_row_val;
	float float_val;
	float running_sum;
	uint64_t row_ind_start;

	// can assume model dim is a multiple of 32...
	unsigned warp_mask = 0xFFFFFFFFU;

	for (int row_id = row_offset; row_id < row_offset + rows_per_block; row_id++){
		row_ind_start = (uint64_t) (row_id) * (uint64_t) n_cols;

		running_sum = 0;

		// 1.) do a per thread loading an initial reduction on max_smem
		for (int i = thread_id; i < n_cols; i+=blockDim.x){
			cur_row_val = X[row_ind_start + i];
			// save for re-scaling
			row[i] = cur_row_val;
			float_val = __half2float(cur_row_val);
			float_val = float_val * float_val;
			running_sum += float_val;
			
		}

		// add this warp's result and place in smem
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			running_sum += __shfl_down_sync(warp_mask, running_sum, warp_offset);
		}

		if (lane_id == 0){
			reduction_data[warp_id] = running_sum;
		}

		__syncthreads();


		// 2.) now combine all the reductions from each thread
		
		if (warp_id == 0){

			running_sum = reduction_data[lane_id];

			for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
				running_sum += __shfl_down_sync(warp_mask, running_sum, warp_offset);
			}

			if (lane_id == 0){
				reduction_data[0] = running_sum;

				// Save down the squared sums of this row
				// so we can easilly compute the backpass...

				// During inference this should be null and not needed
				if (sq_sums){
					sq_sums[row_id] = running_sum;
				}
			}

		}

		__syncthreads();

		
		// now reduction_data[0] has float32 representing total squared sum
		float recip_avg = rsqrtf((reduction_data[0] / (float) n_cols) + eps);

		// 3.) now need to store back all of the row values and mutliply with rms_weight
		__half rms_val;

		for (int i = thread_id; i < n_cols; i+=blockDim.x){
			// copying casting locations as in llama3
			rms_val =  __float2half(__half2float(row[i]) * recip_avg);

			out[row_ind_start + i] = rms_val * weights[i];
		}

		// ensure all threads are complete before we start overwriting row in smem
		__syncthreads();
	}
}

// Because X_inp is in row-major order we should be clever about doing column-wise dot products...

// at the end will do atomicAdds to dW because other blocks will have partial dot products as well

// cannot launch with more threads and n_cols otherwise will be bugs
// # blocks launched is a performance optimization and might be better with less due to less atomicAdds...
// definitely shouldn't launch with more than n_rows
extern "C" __global__ void rms_norm_bwd_weight_fp16_kernel(float eps, int n_rows, int n_cols, __half * X_inp, float * sq_sums, __half * upstream_dX, __half * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// length should be equal to number of rows
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) sdata;

	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) (recip_avgs + n_rows); 

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	for (uint64_t i = thread_id; i < n_rows; i+=blockDim.x){
		recip_avgs[i] = rsqrtf((sq_sums[i] / (float) n_cols) + eps);
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __half2float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * __half2float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2half(weight_derivs[dim]));
	}
}

// Here dX is (N, model_dim) and contains the backprop loss flow that we will update in-place
// This needs to be called after the bwd_weight because the weight we use the updstream dL/dX and this function will
// modify the same pointer...
extern "C" __global__ void rms_norm_bwd_inp_fp16_kernel(float eps, int n_rows, int n_cols, __half * rms_weight, __half * X_inp, float * sq_sums, __half * upstream_dX, __half * dX){
		
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// length should be equal to number of rows
	// load in squared sums and then divide by n_cols and take sqrt
	float * weights_scaled = (float *) sdata;

	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * shared_sq_sums = (float *) (weights_scaled + n_cols); 

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	float dim_scale = rsqrt((float) n_cols); 

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weights_scaled[i] = dim_scale * __half2float(rms_weight[i]);
	}

	// retrieve back the recip squared avgs
	for (uint64_t i = thread_id; i < n_rows; i+=blockDim.x){
		shared_sq_sums[i] = sq_sums[i];
	}

	__syncthreads();

	float deriv;
	float cur_sq_sum;
	float cur_sq_sum_rsqrt;

	float inp_val;

	uint64_t row_ind_start;
	for (int row_id = row_offset; row_id < row_offset + rows_per_block; row_id++){
		row_ind_start = (uint64_t) (row_id) * (uint64_t) n_cols;

		cur_sq_sum = shared_sq_sums[row_id];
		cur_sq_sum_rsqrt = rsqrtf(cur_sq_sum);
		
		for (int i = thread_id; i < n_cols; i+=blockDim.x){
			inp_val = __half2float(X_inp[row_ind_start + i]);
			deriv = (weights_scaled[i] * (cur_sq_sum - (inp_val * inp_val)) * cur_sq_sum_rsqrt) / cur_sq_sum;

			// now update dX
			dX[row_id * n_cols + i] = upstream_dX[row_id * n_cols + i] * __float2half(deriv);

		}
	}
}




// THIS COULD REALLY BE PART OF ATTN KERNEL...
extern "C" __global__ void rope_fp16_kernel(int theta, uint64_t N, int model_dim, int head_dim, int num_kv_heads, int * seq_positions, __half * X_q, __half * X_k) {

	// launched with half the number of threads as output positions because each thread updates two spots
	uint64_t i = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

	// N = total_tokens * model_dim
	if (i < N){



		// ASSUMING model_dim > kv_dim

		// If performance is issue with this divides and modulus
		// we could either use bit tricks or hard-code constants...
		int token_row = i / model_dim;
		int cur_pos = seq_positions[token_row];
		int cur_dim = i % head_dim;

		// probably faster (& simpler) to just use arithmetic functions and recompute
		// instead of loading in from global device memory
		float angle = powf(theta, -1 * ((float) cur_dim / (float) head_dim));
		float cos_val = cosf((float) cur_pos * angle);
		float sin_val = sinf((float) cur_pos * angle);

		float x_even, x_odd; 

		// first do X_q
		x_even = __half2float(X_q[i]);
		x_odd = __half2float(X_q[i + 1]);
		X_q[i] = __float2half(cos_val * x_even - sin_val * x_odd);
		X_q[i + 1] = __float2half(cos_val * x_odd + sin_val * x_even);

		// Now reassign this thread to update x_k
		int kv_dim = num_kv_heads * head_dim;
		token_row = i / (kv_dim);
		int total_tokens = N / model_dim;
		__half i_val;
		__half i_next_val;
		if (token_row < total_tokens){
			cur_pos = seq_positions[token_row];
			cur_dim = i % head_dim;

			angle = powf(theta, -1 * ((float) cur_dim / (float) head_dim));
			cos_val = cosf((float) cur_pos * angle);
			sin_val = sinf((float) cur_pos * angle);

			// now do X_k in same manner but obtaining different x vals
			x_even = __half2float(X_k[i]);
			x_odd = __half2float(X_k[i + 1]);
			i_val = __float2half(cos_val * x_even - sin_val * x_odd);
			i_next_val = __float2half(cos_val * x_odd + sin_val * x_even);

			X_k[i] = i_val;
			X_k[i + 1] = i_next_val;

			// Optimization: Could store in kv cache when already in register instead of 
			// reloading again within kv cache kernel.

			// But the cost of these kernels is minimal compared to the matmuls and attention,
			// so not that big a deal and cleaner to seperate.
		}
	}
}

// N = total_tokens * kv_dim
extern "C" __global__ void copy_kv_to_seq_context_fp16_kernel(uint64_t N, int total_tokens, int kv_dim, __half * keys, __half * values, int * seq_positions, uint64_t * seq_context_ptrs, int * seq_context_sizes){

	uint64_t i = (blockIdx.x * blockDim.x + threadIdx.x);

	if (i < N){

		uint64_t token_ind = i / kv_dim;

		__half * seq_context = (__half *) seq_context_ptrs[token_ind];

		uint64_t seq_pos = seq_positions[token_ind];

		uint64_t cur_dim = i % kv_dim;
		
		seq_context[seq_pos * kv_dim + cur_dim] = keys[token_ind * kv_dim + cur_dim];
		seq_context[(seq_context_sizes[token_ind] * kv_dim) + seq_pos * kv_dim + cur_dim] = values[token_ind * kv_dim + cur_dim];
	}
}


// launching number of blocks determined by seq batch packing
// need to ensure shared memory of at least:
//	- (warp reduction buffers): 32 * 4 
//	- (running output): tokens_in_block * (n_heads/n_kv_heads) * head_dim * 4 
//	- (comp seq pos key+value): head_dim * 4
// 	- (running maxs, half and sums, floats): tokens_in_block * (n_heads/n_kv_heads) * 6


// could consider storing temp output only in halfs to reduce memory by half, but gives up precision (especially for large seqs...)

// aggregate total: 128 + 4 * head_dim + 4 * head_dim * tokens_in_block * (n_heads/n_kv_heads) + 6 * tokens_in_block * (n_heads/n_kv_heads)
//				  : 128 + 4 * head_dim + tokens_in_block * (4 * head_dim * (n_heads/n_kv_heads) + 6 * (n_heads/n_kv_heads))

// max_tokens_in_block => floor((SMEM_SIZE - (128 + 4 * head_dim)) / (4 * head_dim * (n_heads/n_kv_heads) + 6 * (n_heads/n_kv_heads)))

// llama3.1 8B: head_dim = 128 and n_heads/n_kv_heads = 4 => total of: 128 + 512 + 2072 * tokens_in_block bytes
// llama3.1 70B: head_dim = 128 and n_heads/n_kv_heads = 8 => total_of : 128 + 1024 + 8240 * tokens_in_block bytes

// 99KB of shared memory should be available meaning 

// can do this at initialization time
// for Compute Capability 8.6 => need to do: hipFuncSetAttribute(reinterpret_cast<const void*>(cuFunction attn_fp16), hipFuncAttributeMaxDynamicSharedMemorySize, 102400);

// on CC 8.6 for

// q_group_dim = n_heads / n_kv_heads


// for convenience just packing this info into a signle uint64_t array per block
#define BLOCK_CONFIG_MASK_TOKEN_START 0xFFFFFFFF00000000
#define BLOCK_CONFIG_MASK_NUM_TOKENS 0x00000000FFFFFFFF


#define SEQ_PHASE_SIZE 32
#define Q_HEAD_PHASE_SIZE 8

extern "C" __global__ void attention_fp16_kernel(int model_dim, int q_group_dim, int kv_dim, int head_dim, uint64_t * block_configs, int * seq_positions, __half * queries, uint64_t * seq_context_ptrs, int * seq_context_sizes, __half * out) {


	int block_id = blockIdx.y;
	// assume we decide these smartly based on shared memory availability / new tokens per sequenence / overall seq len
	// at least 1 block per sequence in batch

	// this is referring to token index within overall batch

	uint64_t block_info = block_configs[block_id];

	int token_ind_start = (block_info & BLOCK_CONFIG_MASK_TOKEN_START) >> 32;
	int num_tokens_in_block = (block_info & BLOCK_CONFIG_MASK_NUM_TOKENS);
	int block_max_seq_pos = seq_positions[token_ind_start] + num_tokens_in_block;

	// each kernel operates only on a specific kv head
	int kv_head = blockIdx.x;
	int num_threads = blockDim.x;
	int num_warps = num_threads / 32;

	int thread_id = threadIdx.x;
	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	


	/*
	if ((kv_head != 0) || (warp_id != 0) || (block_id != 0)){
		return;
	}
	*/
	

	int num_rows_padded = ROUND_UP_TO_MULTIPLE(num_tokens_in_block * q_group_dim, Q_HEAD_PHASE_SIZE);

	// for every warp in the block will store
	// a temporary reduction buffer. 
	// because max 32 warps per thread block
	// and we can do sync_shfl operations per warp
	// then we can make this by default 32 elements
	// where each warp has a slot


	extern __shared__ uint8_t sdata[];
	__half * block_q_tiles = (__half *) (sdata);
	// keep as float for precision until the end

	__half * block_out_tiles = (__half *) (block_q_tiles + (num_rows_padded * head_dim));


	// in reality this only needs to be one array, but keeping both here for convenience / don't consume much memory
	__half * seq_phase_k_tiles = (__half *) (block_out_tiles + (num_rows_padded * head_dim));
	__half * seq_phase_v_tiles = (__half *) (seq_phase_k_tiles + (head_dim * SEQ_PHASE_SIZE));

	// these will store the current maxes for each row as we progress through
	// sequence iteratively. will be of size q_group_dim
	__half * block_maxs = (__half *) (seq_phase_v_tiles + (head_dim * SEQ_PHASE_SIZE));
	// these will store the current sum for online softmax for each row as we progress
	// through sequence iteratively
	// will keep this as floating point for precision purposes


	// need_num_tokens_in_block * q_group_dim to be a multiple of Q_HEAD_PHASE_SIZE
	float * block_sums = (float *) (block_maxs + num_rows_padded);

	// guaranteed that each thread block will be within one sequence
	__half * seq_keys = (__half *) seq_context_ptrs[token_ind_start];
	// keys and values are stored back to back (but there may be empty
	// space before realloc. we need to now the current size (which is really half of entire cache size
	// in order to offset into values))
	__half * seq_values = seq_keys + kv_dim * seq_context_sizes[token_ind_start];




	const int4 zero_int4 = {0, 0, 0, 0};

	// number of tiles in a grid "row" (which is actually for 8 rows)
	int n_head_tiles = (head_dim >> 3);


	// 1.) init block maxes and block sums

	// num_tokens_in_block * q_group_dim
	for (int cur_ind = thread_id; cur_ind < num_rows_padded; cur_ind+=num_threads){
		// minimum value for fp16
		if (cur_ind < num_tokens_in_block * q_group_dim){
			block_maxs[cur_ind] = NEG_INF_DEV_FP16;
			block_sums[cur_ind] = 0;
		}
		else if (cur_ind < num_rows_padded){
			block_maxs[cur_ind] = CONST_ONE_DEV_FP16;
			block_sums[cur_ind] = 1;
		}
	}

	// 2.) Load q and init output. Likely be more token out spaces then threads.	
	int smem_ind;
	int global_ind;

	int cur_row;
	int tile_base_ind;

	int q_group_dim_bits = __ffs(q_group_dim) - 1;

	// each warp loads tiles for 8 rows
	for (int q_base_row = warp_id * Q_HEAD_PHASE_SIZE; q_base_row < num_rows_padded; q_base_row+=(num_warps * Q_HEAD_PHASE_SIZE)){

		// there are head_dim / 8 tiles per "col"
		// there are 8 rows per "row"
		tile_base_ind = (q_base_row >> 3) * (head_dim >> 3);

		// this is row within the implicit (num tokens in block * q_group_dim, head_dim) matrix
		cur_row = q_base_row + (lane_id & 0x7);

		//token_id = cur_row / (q_group_dim);
		//cur_q_group = cur_row % q_group_dim;
		
		if (cur_row < (num_tokens_in_block * q_group_dim)){

			// loading 8x8 tiles (up to 4 per warp) into unique smem banks
			// each thread loads a row

			// TODO: handle non divisble by 8 edge cases...
			for (int tile_offset = (lane_id >> 3); tile_offset < n_head_tiles; tile_offset+=4){

				
				// x 64 because each tile is 8x8
				// tile_offset increases by 4 each iteration because 4 warps 
				// lane_id >> 2 is row within this tile
				smem_ind = (tile_base_ind + tile_offset) * 64 + (lane_id & 0x7) * 8;

				*((int4 *) &(block_out_tiles[smem_ind])) = zero_int4;

				global_ind = (token_ind_start + (cur_row >> q_group_dim_bits)) * model_dim + kv_head * q_group_dim * head_dim + (cur_row & (q_group_dim - 1)) * head_dim + tile_offset * 8;

				*((int4 *) &(block_q_tiles[smem_ind])) = *((int4 *) &(queries[global_ind]));
			}
		}
		else{

			// loading 8x8 tiles (up to 4 per warp) into unique smem banks
			// each thread loads a row
			for (int tile_offset = (lane_id >> 3); tile_offset < n_head_tiles; tile_offset+=4){
				
				// x 64 because each tile is 8x8
				smem_ind = (tile_base_ind + tile_offset) * 64 + (lane_id & 0x7) * 8;
				
				*((int4 *) &(block_out_tiles[smem_ind])) = zero_int4;
				*((int4 *) &(block_q_tiles[smem_ind])) = zero_int4;
			}

		}
	}

	// __half cur_val;
	__half prev_max;
	__half new_max;
	
	float prev_sum;
	float new_sum;

	__half2 half2_vec;
	const __half head_dim_scale_factor = __float2half(1.0 / sqrtf(head_dim));

	int start_seq_pos = seq_positions[token_ind_start];


	// REGISTERS HOLDING ADDRESS TO LOAD IN MATRIX
	uint32_t RA[4];
	uint32_t RB[4];
	uint32_t RC[2];
	// used to get the output of transpose before writing to smem
	uint32_t RTemp[2];

	uint32_t A_lane_addr;
	uint32_t B_lane_addr;
	uint32_t C_lane_addr;

	int cur_base_q_row;
	int cur_base_token_pos;

	int max_seq_offset;

	//unsigned warp_mask;


	for (int cur_base_seq_phase = 0; cur_base_seq_phase < block_max_seq_pos; cur_base_seq_phase+=SEQ_PHASE_SIZE) {

		// ensure to sync before loading new seq phase for kvs...
		__syncthreads();

		for (int seq_base_row = warp_id; seq_base_row < SEQ_PHASE_SIZE; seq_base_row+=num_warps) {
			
			tile_base_ind = (seq_base_row >> 3) * (head_dim >> 3);

			if ((cur_base_seq_phase + seq_base_row) < block_max_seq_pos){
				for (int tile_offset = lane_id; tile_offset < n_head_tiles; tile_offset+=32){
					
					smem_ind = (tile_base_ind + tile_offset) * 64 + ((seq_base_row + tile_offset) & 0x7) * 8;

					global_ind = (cur_base_seq_phase + seq_base_row) * kv_dim + kv_head * head_dim + tile_offset * 8;

					*((int4 *) &(seq_phase_k_tiles[smem_ind])) = *((int4 *) &(seq_keys[global_ind]));
					*((int4 *) &(seq_phase_v_tiles[smem_ind])) = *((int4 *) &(seq_values[global_ind]));;

				}
			}
			else{
				for (int tile_offset = lane_id; tile_offset < n_head_tiles; tile_offset+=32){
					smem_ind = (tile_base_ind + tile_offset) * 64 + ((seq_base_row + tile_offset) & 0x7) * 8;
					*((int4 *) &(seq_phase_k_tiles[smem_ind])) = zero_int4;
					*((int4 *) &(seq_phase_v_tiles[smem_ind])) = zero_int4;
				}

			}
		}

		__syncthreads();

		// Seq phases are 32 x head dim matrices 


		// remember this is for this specific kv head, which is head_dim entries per position



		// TL;DR Each warp takes down 8 rows of "Q", total number of rows is #(tokens in block * q_group_dim
		// The output goes in a temporary buffer that gets updated every phase 
		// which is then finally put into output array after all the sequence phases. Tokens in block is assigned
		// during seq batch finalization and is paired with the launch configuration. Every block must have
		// tokens only in the same sequence and they must be ordered in monitonically increasing sequence positions.


		// Each warp within an iteration (responsible for a specific output chunk) will do a series of <head_dim / k> matrix multiples each of (8, 16) x (16, 32)
		// in order to obtain the correct dot product for 8 "rows" of Q. 
		// Note: Every token will produce q_group_dim rows in this generalized Q matrix correspondign to this kv head.
		// but tokens can be aggrated if they are part of the same sequence, so a given warp iteration during this phase 
		// might update the a chunk of parital outputs of 2 tokens if  q_group_dim = 4 (llama 8B) or 1 token if (q_group_dim = 8). 
		// Here we are taking advantage of the fact that different query heads utilize 
		// the same keys/values so we are grouping them in matmul form.


		// The output of the dot-product phase yields a 8x32 matrix where the columns are dot product of the given query associated with current
		// "row of Q" with the 32 sequences during this phase. (In reality we write the result in column major format for good
		// reading of the tile during the matmul's with values). We then do online softmax across each of these rows (continuation
		// of previous seq phases). 

		// After softmax is completed we and properly update the output by doing 2 sets (because 32 / 16 = 2) 
		// of <head_dim / 32> matmuls of size (8, 16) x (16, 32) where the former matrix is 16 sequences out of 32 across each of the 8
		// rows this warp is working with. The latter matrix is a porition of the values assoicated with these 16 sequence poisitions and 
		// here we are grabbing 32 entries out of the head dim. This processess is repeated if each warp needs to process more 
		// than 8 rows of Q in order to satisify processed all the tokens that have been assigned to this threadblock. 


		


		// The number of iterations each warp will work is dependent on how many total "rows" of Q are in this matrix
		// (corresponding to this threadblock) and how many total warps there are. The mapping between real token rows
		// and how many total tokens each threadblock should work on is configurated during runtime because
		// (hardware constrains, dimension of model, sequence lengths in batch) all play a role. Note that every threadblock only works
		// on tokens that all are part of the same sequence and there will be as many threadblocks assigned to sequence as is necessary
		// to process all of the new tokens part of the sequence.


		// BIG TODO: make a queue so warps can be more balanced instead of waiting at the loading barrier...


		cur_base_q_row = warp_id * Q_HEAD_PHASE_SIZE;
		cur_base_token_pos = start_seq_pos + (cur_base_q_row >> q_group_dim_bits);

		// advance past tokens that have already completed (i.e. they don't need to compare against this seq phase because of masking)
		while (cur_base_token_pos < cur_base_seq_phase){
			cur_base_q_row += num_warps * Q_HEAD_PHASE_SIZE;
			cur_base_token_pos += num_warps * (Q_HEAD_PHASE_SIZE >> q_group_dim_bits);
		}

		while (cur_base_q_row < (num_tokens_in_block * q_group_dim)) {


			// 1.) get QK^T

			// We have that Q is (num_tokens_in_block * head_dim) x head_dim matrix and that K is a 32xhead_dim matrix

			// This warp is currently operating on a base row of Q that should be (8 x head_dim)
			// We will do successive matmuls to then get an (8 x 32) intermeidate matrix that we can do online softmax on

			// However we want to use m16, k16, n8 MMA instruction so in reality, we will be doing KQ^T == QK^T
			// meaning our output will be transposed into (32xhead_dim), which is what we want for softmax phase

			// Because seq phase is 32, we will do 2 rounds


			// The A matrix comes from seq_phase_k and takes in 4 registers corresponding to the 4 different 8x8 sub-matrices that make up 16x16 A matrix
			// Each group of 4 threads loads 16 bytes == 8 elements == 1 row

			// The rows of the first matrix are specified by the addresses provided by threads 0-7
			// The rows of the second matrix are specified by the addresses provided by threads 8-15
			// The rows of the third matrix are specified by the addresses provided by threads 16-23
			// The rows of the fourth matrix are specified by the addresses provided by threads 16-23


			tile_base_ind = (cur_base_q_row >> 3) * (head_dim >> 3);

			half2_vec = __half2half2(head_dim_scale_factor);

			#pragma unroll
			for (int s = 0; s < SEQ_PHASE_SIZE / 16; s++){

				
				// REF: https://docs.nvidia.com/cuda/pdf/ptx_isa_8.5.pdf (pages 398 & page 364)

				// Initialize the output to zero
				RC[0] = 0;
				RC[1] = 0;

				
				// doing 2 head tiles per iteration because (16, 16) x (16, 8) matmul
				for (int k = 0; k < n_head_tiles; k+=2){




					// doing (16, 16) x (16, 8) matmul where A is keys and B is queries



					// s * 16 indicates if we are in the top half of the 32xhead dim or bottom half

					// lane_id & 0x8 indicates if threads are in groups 8-15 or 24-31 (in which case needs to load in grid offset)
					// lane_id & 0x7 indicates the row within 8x8 tile

					// lane_id >> 4 indicates if threads are in groups 16-23 or 24-31 in which case they need to offset by 8 elements in row = 1 tile
				

					// we permuted the rows in each tile to allow for conflict free so using ((lane_id + k + (lane_id >> 4)) & 0x7) to obtain true row
					smem_ind = ((s * (SEQ_PHASE_SIZE / 16) * n_head_tiles + ((lane_id & 0x8) >> 3) * n_head_tiles + k + (lane_id >> 4)) * 64) + ((lane_id + k + (lane_id >> 4)) & 0x7) * 8;
					A_lane_addr = __cvta_generic_to_shared(&(seq_phase_k_tiles[smem_ind]));
					
					// Spliting the 16x16 matrix into 4 submatices

					// RA[0] is top left, RA[1] is bottom left, RA[2] is bottom left, and RA[3] is bottom right
					// (this is based off the thread addressing scheme)

					LDMATRIX_X4(RA[0], RA[1], RA[2], RA[3], A_lane_addr);


					// block_q is (num_padded_rows x head_dim) matrix, but we only want to load in the 8 rows following cur_base_q_row

					// upper 16 lanes don't matter

					// using tile_base_ind derived from cur_base_q row

					// (lane_id & 0x8) >> 3 indicates if this thread should load from a tile offset
					smem_ind = (tile_base_ind + k + ((lane_id & 0x8) >> 3)) * 64 + (lane_id & 0x7) * 8;
					B_lane_addr = __cvta_generic_to_shared(&(block_q_tiles[smem_ind]));

					// RB[0] will then contain an 8x8 matrix where each row has 8 dims and each column corresponds to row in q matrix
					// RB[1] contains the second string of 8 dims and each column corerspondds to row in q matrix

					// i.e. each address should be referring to a sequence of 8 elements that represent half of a column
					LDMATRIX_X2(RB[0], RB[1], B_lane_addr);

					

					// B is expected to be in column-major format as (16x8)
					HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[0], RB[1], RC[0], RC[1]);
				}

				

				
				// Now we have the full dot product for these 16 sequences and we can save the results to seq phase out
				// in order to do online softmax

				// Save this section of 16 sequenc

				// each thread holds 2 f16x2 outputs (which are packed into u32 register) of the 16x8 result, offset by 8 rows in the 16x8 output


				// storing in tranposed order so we have a 8 x 16 matrix represetnting QK^T as we should...

				// MATMOV ptx instruction
				MAT_TRANS(RTemp[0], RC[0]);
				MAT_TRANS(RTemp[1], RC[1]);



				// get the correct row, but then cast to uint32_t and offset by lane_id % 4 as uint32_t's because each thread holds 2 fp16 items

				// scale by head dim

				if (s == 1){
					*((__half2 *)(&RA[2 * s])) = __hmul2(*(__half2 *) (&RTemp[0]), half2_vec);
    				*((__half2 *)(&RA[2 * s + 1])) = __hmul2(*(__half2 *) (&RTemp[1]), half2_vec);
    			}
    			else{
    				*((__half2 *)(&RB[2])) = __hmul2(*(__half2 *) (&RTemp[0]), half2_vec);
    				*((__half2 *)(&RB[3])) = __hmul2(*(__half2 *) (&RTemp[1]), half2_vec);
    			}
			}

			// Now copy RB[2] and RB[3] to RA
			// (RA[2] and RA[3] were updated in the last iteration so they are already correct)
			RA[0] = RB[2];
			RA[1] = RB[3];

			// We will need to save the previous max and sum when modifying the outputs
			// Each lane has a designated row of outputs to update and they will update 4 elements
			// each within the 8x16 matrix (2 from first 8x8 and 2 from second 8x8)
			prev_max = block_maxs[cur_base_q_row + (lane_id >> 2)];
			prev_sum = block_sums[cur_base_q_row + (lane_id >> 2)];
			
			// Apply causal mask
			max_seq_offset = min(cur_base_token_pos + ((lane_id >> 2) >> q_group_dim_bits) - cur_base_seq_phase, SEQ_PHASE_SIZE);
			
			// CAUSAL MASK
			new_max = prev_max;

			for (int seq_section = 0; seq_section < 4; seq_section++){

				if (((seq_section << 3) + 2 * (lane_id & 0x3) + 1) <= max_seq_offset){
					new_max = __hmax(new_max, __hmax(__low2half(*((__half2 *) &RA[seq_section])),  __high2half(*((__half2 *) &RA[seq_section]))));
				}
				else if (((seq_section << 3) + 2 * (lane_id & 0x3)) == max_seq_offset){
					new_max = __hmax(new_max, __low2half(*((__half2 *) &RA[seq_section])));
					*((__half2 *) (&RA[seq_section])) = __halves2half2(__low2half(*((__half2 *) &RA[seq_section])), NEG_INF_DEV_FP16);
				}
				else{
					*((__half2 *) (&RA[seq_section]))  = __halves2half2(NEG_INF_DEV_FP16, NEG_INF_DEV_FP16);
				}
			}

			new_max = __hmax(new_max, __shfl_down_sync(0xFFFFFFFF, new_max, 2, 4));
			new_max = __hmax(new_max, __shfl_down_sync(0xFFFFFFFF, new_max, 1, 4));
			
			new_max = __shfl_sync(0xFFFFFFFF, new_max, 0, 4);

			new_sum = 0;
			
			float v1;
			float v2;

			// already set maked value to neg inf, so no need to check again
			#pragma unroll
			for (int seq_section = 0; seq_section < 4; seq_section++){
				v1 = expf(__half2float(__low2half(*((__half2 *) &RA[seq_section])) - new_max));
				v2 = expf(__half2float(__high2half(*((__half2 *) &RA[seq_section])) - new_max));
				new_sum += v1 + v2;
				*((__half2 *) (&RA[seq_section])) = __halves2half2(__float2half(v1), __float2half(v2));
			}

			new_sum += __shfl_down_sync(0xFFFFFFFF, new_sum, 2, 4);
			new_sum += __shfl_down_sync(0xFFFFFFFF, new_sum, 1, 4);

			// now the results are lanes where lane_id & 0x3 == 0
			if ((lane_id & 0x3) == 0){
				new_sum += prev_sum * expf(__half2float(prev_max - new_max));
				block_maxs[cur_base_q_row + (lane_id >> 2)] = new_max;
				// update block sums/maxs
				block_sums[cur_base_q_row + (lane_id >> 2)] = new_sum;
			}

			// loading value from the row leader...
			__syncwarp();

			new_sum = block_sums[cur_base_q_row + (lane_id >> 2)];
			new_max = block_maxs[cur_base_q_row + (lane_id >> 2)];

			// now update B matrix using A matrix modified (masked out/adjusted attention scores)
			#pragma unroll
			for (int seq_section = 0; seq_section < 4; seq_section++){
				*((__half2 *) (&RB[seq_section])) = __h2div(*((__half2 *) &RA[seq_section]), __half2half2(__float2half(new_sum)));
			}



			// Now seq_phase_out contains an 8x32 matrix of attention scores for this set of 32 sequences
			// across the 8 rows of q

			// We need to do matmul with values corresponding to these sequences and update the temporary output
			// (held in block_out)

			// Similarly to our KQ^T computation we can do V^TS^T where V is (head_dim, 32) and S^T is (32, 8)

			// The result is then a (head_dim, 8) portion of outputs for the 8 rows

			// We can do head_dim / 16 outer rounds for portion of head dim, and then 2 inner rounds for each half of the seq phase

			// However we now are not starting from scratch but rather loading the previous output, doing scalar updates per row
			// and the adding the new results

			// the row id to update for both matrices is lane_id >> 2 (= lane_id / 4)
			// scale based on new max and new sum relative to prev sum and prev max
			// this thread already saved the prev max and prev sum before the new ones were calculated during softmax
			half2_vec = __half2half2(__float2half((prev_sum / new_sum) * expf(__half2float(prev_max - new_max))));

			for (int k = 0; k < n_head_tiles; k+=2) {

				// a.) Load prior outputs (8x16) along this head dim
				
				// block out is (num_tokens_in_block * q_group_dim) x head_dim

				// we want to take a slice of [cur_base_q_row: cur_base_q_row + 8, k * 16: k * 16 + 16]
							
				// same indexing scheme as in block_q
				// upper 16 lanes don't matter
				
				// lanes 0-7 are going to load tile # k
				// lanes 8-15 are going to load tile # k + 1

				smem_ind = (tile_base_ind + k + ((lane_id & 0x8) >> 3)) * 64 + (lane_id & 0x7) * 8;
				C_lane_addr = __cvta_generic_to_shared(&(block_out_tiles[smem_ind]));
				LDMATRIX_X2(RTemp[0], RTemp[1], C_lane_addr);

				// RC[0] contains an 8x8 matrix with the where each row correspnds to a row of Q and the columns are the first 8 elements of head dim
				// RC[1] contains 8x8 matrix with the second portion of 8 head dim els
	

				// need to multiply the 4 half elements in RC[0] and RC[1] by this amount
				// probably an instrinstic to do this cleaner...
				*((__half2 *) (&RTemp[0])) = (__hmul2((*((__half2 *) &RTemp[0])), half2_vec));
				*((__half2 *) (&RTemp[1])) = (__hmul2((*((__half2 *) &RTemp[1])), half2_vec));
				

				// TODO: For convience with output storing in row major
				// but really should have block out be column major during computation
				// and then at the final output convert back (to avoid these transposes...)
				// also would need to modify how thread out scale chooses the elements to modify...

				MAT_TRANS(RC[0], RTemp[0]);
				MAT_TRANS(RC[1], RTemp[1]);

				#pragma unroll
				for (int s = 0; s < SEQ_PHASE_SIZE / 16; s++){
					
					// now need to load a 16x16 chunk of values

					// where the values are held within (32, head dim) matrix

					// we want to get 4 8x8 matrices (16x16) where the rows are unique head dim and columns are unique seq inds
					// thus we want to transpose

					// the top left matrix when we do matmul (RA[0]) should be the (first 8 head_dim x first 8 sequences). The bottom left 
					// matrix when we do matmul (RA[1]) should be (second set of 8 head_dim x first 8 sequences) and the top right
					// (RA[2]) should be (first 8 head_dim x second set of 8 sequences)

					// Addresses from lanes 0-7 correspond to rows of RA[0], from lanes 8-15 correspond to RA[1], etc.


					// same as loading from seq_pahse_k_tiles...
					smem_ind = ((s * (SEQ_PHASE_SIZE / 16) * n_head_tiles + ((lane_id & 0x8) >> 3) * n_head_tiles + k + (lane_id >> 4)) * 64) + ((lane_id + k + (lane_id >> 4)) & 0x7) * 8;
					A_lane_addr = __cvta_generic_to_shared(&(seq_phase_v_tiles[smem_ind]));

					// RA[0] is top left, RA[1] is bottom left, RA[2] is bottom left, and RA[3] is bottom right
					// (this is based off the thread addressing scheme)

					// However because we are transposing each 8x8 matrix we want to switch RA[2] and RA[1]
					// so that RA[1] and RA[2] point to the lower left and upper right of transposed 16x16 matrix
					// respectively...
					// (and will multiply them normal later)
					LDMATRIX_X4_TRANS(RA[0], RA[2], RA[1], RA[3], A_lane_addr);


					// // and load in the seq phase out portion
					// // only lower 16 threads pass in addresses here

					// // we want the first 8 threads to load in the first 8 sequences
					// // and second to load the next 8 sequences

					// // the (lane_id + (2 * s + ((lane_id & 0x8) >> 3))) & 0x7) is because within seq phase out we are stroing permuted rows to avoid bank conflicts during softmax
					// B_lane_addr =  __cvta_generic_to_shared(&(seq_phase_out_tiles[warp_id * (SEQ_PHASE_SIZE * Q_HEAD_PHASE_SIZE) + (2 * s + ((lane_id & 0x8) >> 3)) * 64 + ((lane_id + (2 * s + ((lane_id & 0x8) >> 3))) & 0x7) * 8]));
					// LDMATRIX_X2(RB[0], RB[1], B_lane_addr);

					// We have already loaded B into 4 registers, and we will choose the correct 2 based
					// on seq phase

					HMMA16816(RC[0], RC[1], RA[0], RA[1], RA[2], RA[3], RB[2 * s], RB[2 * s + 1], RC[0], RC[1]);

				}

			
				MAT_TRANS(RTemp[0], RC[0]);
				MAT_TRANS(RTemp[1], RC[1]);


				// get the correct row, but then cast to uint32_t and offset by lane_id % 4 as uint32_t's because each thread holds 2 fp16 items
				*((uint32_t *)(&block_out_tiles[(tile_base_ind + k) * 64 + (lane_id >> 2) * 8]) + (lane_id & 0x3)) = RTemp[0];
				// the next set of 8 columns
    			*((uint32_t *)(&block_out_tiles[(tile_base_ind + k + 1) * 64 + (lane_id >> 2) * 8]) + (lane_id & 0x3)) = RTemp[1];

			}

			// repeat for other tokens
			cur_base_q_row += num_warps * Q_HEAD_PHASE_SIZE;
			cur_base_token_pos += num_warps * (Q_HEAD_PHASE_SIZE >> q_group_dim_bits);
		}
	}


	// Wrapped up and time to do output...
	__syncthreads();

	// each warp loads tiles for 8 rows
	for (int q_base_row = warp_id * Q_HEAD_PHASE_SIZE; q_base_row < num_rows_padded; q_base_row+=(num_warps * Q_HEAD_PHASE_SIZE)){

		// there are head_dim / 8 tiles per "col"
		// there are 8 rows per "row"
		tile_base_ind = (q_base_row >> 3) * (head_dim >> 3);

		// this is row within the implicit (num tokens in block * q_group_dim, head_dim) matrix
		cur_row = q_base_row + (lane_id & 0x7);

		//token_id = cur_row / (q_group_dim);
		//cur_q_group = cur_row % q_group_dim;
		
		if (cur_row < (num_tokens_in_block * q_group_dim)){

			// loading 8x8 tiles (up to 4 per warp) into unique smem banks
			// each thread loads a row

			// TODO: handle non divisble by 8 edge cases...
			for (int tile_offset = (lane_id >> 3); tile_offset < n_head_tiles; tile_offset+=4){

				global_ind = (token_ind_start + (cur_row >> (q_group_dim_bits))) * model_dim + kv_head * q_group_dim * head_dim + (cur_row & (q_group_dim - 1)) * head_dim + tile_offset * 8;

				// x 64 because each tile is 8x8
				// tile_offset increases by 4 each iteration because 4 warps 
				// lane_id >> 2 is row within this tile
				smem_ind = (tile_base_ind + tile_offset) * 64 + (lane_id & 0x7) * 8;

				*((int4 *) &(out[global_ind])) = *((int4 *) &(block_out_tiles[smem_ind]));
			}
		}
	}
}



extern "C" __global__ void silu_hadamard_fp16_kernel(uint64_t N, __half * x_w1, __half * x_w3, __half * out){

	uint64_t i = (blockIdx.x * blockDim.x + threadIdx.x);
	// here N is total_tokens * ffn_dim 
	if (i < N){

		float x_w1_val = __half2float(x_w1[i]);
		float x_w3_val = __half2float(x_w3[i]);

		// overwrite contents in x_w1
		float silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));
		
		// normally would set out to be x_w1...
		out[i] = __float2half(silu_x_w1 * x_w3_val);
	}
}


extern "C" __global__ void condense_rows_fp16_kernel(uint64_t N, int n_rows, int n_cols, __half * X_in, __half * X_out, int * row_remapping) {

	uint64_t i = (blockIdx.x * blockDim.x + threadIdx.x);
	if (i < N){
		int prev_row = i / n_cols;

		int new_row = row_remapping[prev_row];
		if (new_row == -1){
			return;
		}

		int cur_col = i % n_cols;
		
		__half cur_val = X_in[i];

		X_out[new_row * n_cols + cur_col] = cur_val;
	}
}




// Assumes N = # columns
// And block_idx is the row

// very naive implementation for now....
extern "C" __global__ void softmax_fp16_to_float_kernel(int n_cols, __half * X_in, float * out, uint32_t * arg_maxs) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__half * row_start = X_in + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __half warp_maxs[32];
	__shared__ __half warp_sums[32];
	__shared__ __half global_max[1];
	__shared__ __half global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_FP16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__half other_val;

	__half new_max = NEG_INF_DEV_FP16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, row_start[cur_ind]);

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__half overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	__half cur_val;
	while (cur_ind < n_cols){

		cur_val = row_start[cur_ind];
		if (arg_maxs && (cur_val == overall_max)){
			arg_maxs[row_ind] = (uint32_t) cur_ind;
		}

		new_sum = expf(__half2float(cur_val - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	float * out_start = out + row_offset;

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_start[cur_ind] = expf(__half2float(row_start[cur_ind] - overall_max)) / overall_sum;
		cur_ind += num_warps * 32;
	}
}

// TODO: could read in row of data to smem...
extern "C" __global__ void softmax_fp16_kernel(int n_cols, __half * X) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__half * row_start = X + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __half warp_maxs[32];
	__shared__ __half warp_sums[32];
	__shared__ __half global_max[1];
	__shared__ __half global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_FP16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__half other_val;

	__half new_max = NEG_INF_DEV_FP16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, row_start[cur_ind]);

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__half overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__half2float(row_start[cur_ind] - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		row_start[cur_ind] = __float2half(expf(__half2float(row_start[cur_ind] - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}

// subtracts 1 from correct values

// launched with number of rows (total tokens to predict)
extern "C" __global__ void cross_entropy_loss_fp16_kernel(int n_rows, int n_cols, __half * pred_logits, uint32_t * labels){

	uint64_t i = (blockIdx.x * blockDim.x + threadIdx.x);

	int row_ind;
	uint32_t correct_ind;
	if (i < n_rows){
		row_ind = i / n_cols;
		correct_ind = labels[row_ind];
		pred_logits[row_ind * n_cols + correct_ind] -= CONST_ONE_DEV_FP16;
	}

}
